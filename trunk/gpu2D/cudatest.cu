#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipfft/hipfft.h>

extern "C" void TESTCUDA(void)
{
void *pt;
  hipMalloc(&pt, 100000);
  hipFree(pt);
}

typedef float2 Complex;

int c_numx, c_numy;
float c_elch, c_beta;
Complex c_k0, c_k1;
float *c_tmpbuf;
float *g_pot, *g_pot_file_static, *g_pot_filelist, *g_potx, *g_poty, *g_Kx, *g_Ky;
Complex *g_psi, *g_psik;
hipfftHandle g_fft_hand;

void double_to_float(float *fpt, double *dpt, int n)
{
int i;

  for (i = 0; i < n; i++)
    fpt[i] = (float)dpt[i];
}

void float_to_double(double *dpt, float *fpt, int n)
{
int i;

  for (i = 0; i < n; i++)
    dpt[i] = (double)fpt[i];
}

int iDivUp(int a, int b)
{
  return (a % b != 0) ? (a / b + 1) : (a / b);
}

__global__ void
add_potential(int numx, int numy, float *pot, float *pot_file_static, float *pot_filelist, float *potx, float *poty, float ELCH)
{
  const int ix = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
  const int iy = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
  int idx = mul24(ix, numy) + iy;

  pot[idx] = (pot[idx] + pot_file_static[idx] + pot_filelist[idx] + potx[ix] + poty[iy])*ELCH;
}

__global__ void
nonlinear_half_step(int numx, int numy, Complex *psi, float *pot, Complex k0, float beta)
{
  const int ix = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
  const int iy = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
  int idx = mul24(ix, numy) + iy;
  Complex cpl, tmp;
  float v1, v2;
  v1 = pot[idx] + beta*(psi[idx].x*psi[idx].x + psi[idx].y*psi[idx].y);
  v2 = exp(v1*k0.x);
  cpl.x = v2 * cos(v1*k0.y);
  cpl.y = v2 * sin(v1*k0.y);
  tmp.x = psi[idx].x*cpl.x - psi[idx].y*cpl.y;
  tmp.y = psi[idx].y*cpl.x + psi[idx].x*cpl.y;
  psi[idx] = tmp;
}

__global__ void
nonlinear_half_step_scaled(int numx, int numy, Complex *psi, float *pot, Complex k0, float beta, float scaling)
{
  const int ix = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
  const int iy = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
  int idx = mul24(ix, numy) + iy;
  Complex cpl, tmp;
  float v1, v2;
  v1 = pot[idx] + beta*(psi[idx].x*psi[idx].x + psi[idx].y*psi[idx].y);
  v2 = scaling*exp(v1*k0.x);
  cpl.x = v2 * cos(v1*k0.y);
  cpl.y = v2 * sin(v1*k0.y);
  tmp.x = psi[idx].x*cpl.x - psi[idx].y*cpl.y;
  tmp.y = psi[idx].y*cpl.x + psi[idx].x*cpl.y;
  psi[idx] = tmp;
}

__global__ void
linear_step(int numx, int numy, Complex *psi, float *kx, float *ky, Complex k1)
{
  const int ix = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
  const int iy = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
  int idx = mul24(ix, numy) + iy;
  Complex cpl, tmp;
  float v1, v2;
  v1 = kx[ix] + ky[iy];
  v2 = exp(v1*k1.x);
  cpl.x = v2 * cos(v1*k1.y);
  cpl.y = v2 * sin(v1*k1.y);
  tmp.x = psi[idx].x*cpl.x - psi[idx].y*cpl.y;
  tmp.y = psi[idx].y*cpl.x + psi[idx].x*cpl.y;
  psi[idx] = tmp;
}

extern "C" void GPUSPLIT_INIT(int *numx, int *numy, double *elch, double *k0_real, double *k0_img,
		double *k1_real, double *k1_img, double *beta, double *Kx, double *Ky,
		double *pot_file_static, double *psi)
{
  c_numx = *numx;
  c_numy = *numy;
  c_elch = (float)(*elch);
  c_k0.x = (float)(*k0_real);
  c_k0.y = (float)(*k0_img);
  c_k1.x = (float)(*k1_real);
  c_k1.y = (float)(*k1_img);
  c_beta = (float)(*beta);
  c_tmpbuf = (float *)malloc(4*c_numx*c_numy*sizeof(float));
  hipMalloc(&g_pot, 4*c_numx*c_numy);
  hipMalloc(&g_pot_file_static, 4*c_numx*c_numy);
  hipMalloc(&g_pot_filelist, 4*c_numx*c_numy);
  hipMalloc(&g_potx, 4*c_numx);
  hipMalloc(&g_poty, 4*c_numy);
  hipMalloc(&g_psi, 2*4*c_numx*c_numy);
  hipMalloc(&g_Kx, 4*c_numx);
  hipMalloc(&g_Ky, 4*c_numy);
  
  double_to_float(c_tmpbuf, Kx, c_numx);
  hipMemcpy(g_Kx, c_tmpbuf, c_numx*sizeof(float), hipMemcpyHostToDevice);
  double_to_float(c_tmpbuf, Ky, c_numy);
  hipMemcpy(g_Ky, c_tmpbuf, c_numy*sizeof(float), hipMemcpyHostToDevice);
  double_to_float(c_tmpbuf, pot_file_static, c_numx*c_numy);
  hipMemcpy(g_pot_file_static, c_tmpbuf, c_numx*c_numy*sizeof(float), hipMemcpyHostToDevice);
  double_to_float(c_tmpbuf, psi, 2*c_numx*c_numy);
  hipMemcpy(g_psi, c_tmpbuf, 2*c_numx*c_numy*sizeof(float), hipMemcpyHostToDevice);
  
  hipMemset(g_pot_filelist, 0, 4*c_numx*c_numy);
  
  hipfftPlan2d(&g_fft_hand, c_numx, c_numy, HIPFFT_C2C);
}

extern "C" void GPUSPLIT_DO_STEP(double *pot, double *pot_filelist, double *potx, double *poty, int *filelist_changed)
{
  double_to_float(c_tmpbuf, pot, c_numx*c_numy);
  hipMemcpy(g_pot, c_tmpbuf, c_numx*c_numy*sizeof(float), hipMemcpyHostToDevice);
  if (*filelist_changed)
  {
    double_to_float(c_tmpbuf, pot_filelist, c_numx*c_numy);
    hipMemcpy(g_pot_filelist, c_tmpbuf, c_numx*c_numy*sizeof(float), hipMemcpyHostToDevice);
  }
  double_to_float(c_tmpbuf, potx, c_numx);
  hipMemcpy(g_potx, c_tmpbuf, c_numx*sizeof(float), hipMemcpyHostToDevice);
  double_to_float(c_tmpbuf, poty, c_numy);
  hipMemcpy(g_poty, c_tmpbuf, c_numy*sizeof(float), hipMemcpyHostToDevice);

  dim3 threadBlock(16, 16);
  dim3 kernelBlockGrid(iDivUp(c_numx, threadBlock.x), iDivUp(c_numy, threadBlock.y));
  add_potential<<<kernelBlockGrid, threadBlock>>>(c_numx, c_numy, g_pot, g_pot_file_static, g_pot_filelist, g_potx, g_poty, c_elch);

  nonlinear_half_step<<<kernelBlockGrid, threadBlock>>>(c_numx, c_numy, g_psi, g_pot, c_k0, c_beta);
  hipfftExecC2C(g_fft_hand, (hipfftComplex *)g_psi, (hipfftComplex *)g_psi, HIPFFT_FORWARD);
  linear_step<<<kernelBlockGrid, threadBlock>>>(c_numx, c_numy, g_psi, g_Kx, g_Ky, c_k1);
  hipfftExecC2C(g_fft_hand, (hipfftComplex *)g_psi, (hipfftComplex *)g_psi, HIPFFT_BACKWARD);

  nonlinear_half_step_scaled<<<kernelBlockGrid, threadBlock>>>(c_numx, c_numy, g_psi, g_pot, c_k0, c_beta, 1.0F/((float)(c_numx*c_numy)));
}

extern "C" void GPUSPLIT_GET_PSI(double *psi)
{
  hipMemcpy(c_tmpbuf, g_psi, 2*c_numx*c_numy*sizeof(float), hipMemcpyDeviceToHost);
  float_to_double(psi, c_tmpbuf, 2*c_numx*c_numy);
}

extern "C" void GPUSPLIT_GET_POT(double *pot)
{
  hipMemcpy(c_tmpbuf, g_pot, c_numx*c_numy*sizeof(float), hipMemcpyDeviceToHost);
  float_to_double(pot, c_tmpbuf, c_numx*c_numy);
}

extern "C" void GPUSPLIT_DESTROY(void)
{
  free(c_tmpbuf);
  hipFree(&g_pot);
  hipFree(&g_pot_file_static);
  hipFree(&g_pot_filelist);
  hipFree(&g_potx);
  hipFree(&g_poty);
  hipFree(&g_psi);
  hipFree(&g_Kx);
  hipFree(&g_Ky);
  hipfftDestroy(g_fft_hand);
}
